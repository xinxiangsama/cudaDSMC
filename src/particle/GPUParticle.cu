#include "hip/hip_runtime.h"
#include "Particle.cuh"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>


GPUParticles::GPUParticles(const int &particleNum) : N(particleNum)
{   
    size_t sizedoubles {N * sizeof(double)};   
    size_t sizedouble3s {N * sizeof(double3)};
    size_t sizeints {N * sizeof(int)};
    hipMalloc((void**)&d_mass, sizedoubles);
    hipMalloc((void**)&d_pos, sizedouble3s);
    hipMalloc((void**)&d_vel, sizedouble3s);
    hipMalloc((void**)&global_id, sizeints);
	hipMalloc((void**)&global_id_sortted, sizeints);
    hipMalloc((void**)&cell_id, sizeints);
    hipMalloc((void**)&local_id, sizeints);
    hipMalloc((void**)&d_injectedCounter, sizeof(int));

}

GPUParticles::~GPUParticles()
{
    hipFree(d_mass);
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(global_id);
	hipFree(global_id_sortted);
    hipFree(local_id);
    hipFree(cell_id);
    hipFree(d_injectedCounter);
}

void GPUParticles::UploadFromHost(const double* h_mass,
    const double3* h_pos,
    const double3* h_vel, 
    const int* h_global_id, const int* h_local_id, const int* h_cell_id)
{
    size_t sizedoubles {N * sizeof(double)};   
    size_t sizedouble3s {N * sizeof(double3)};
    size_t sizeints {N * sizeof(int)};
    hipMemcpy(d_mass, h_mass, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_pos, h_pos, sizedouble3s, hipMemcpyHostToDevice);
    hipMemcpy(d_vel, h_vel, sizedouble3s, hipMemcpyHostToDevice);

    hipMemcpy(global_id, h_global_id, sizeints, hipMemcpyHostToDevice);
    hipMemcpy(local_id, h_local_id, sizeints, hipMemcpyHostToDevice);
    hipMemcpy(cell_id, h_cell_id, sizeints, hipMemcpyHostToDevice);
}

void GPUParticles::Move(const double &dt, const double &blockSize, const Boundary* d_boundaries,
    const int* d_ifCut, const GPUSegment* d_Segments)
{   
    int* d_valid;
    hipMalloc((void**)&d_valid, N * sizeof(int));
    hipMemset(d_valid, 1, N * sizeof(int));
    int numBlocks = (N + blockSize - 1) / blockSize;
    GPUParticleKernels::moveParticles<<<numBlocks, blockSize>>>(d_pos,
                                          d_vel,
                                          N, dt, d_boundaries,
                                          d_ifCut, d_Segments, cell_id, d_valid);
    hipDeviceSynchronize();
    DeleteInvalid(d_valid);
    hipFree(d_valid);
}

void GPUParticles::DeleteInvalid(int* d_valid) {

    // 创建 Thrust 设备指针
    thrust::device_ptr<double3> pos_ptr(d_pos);
    thrust::device_ptr<double3> vel_ptr(d_vel);
    thrust::device_ptr<double> mass_ptr(d_mass);
    thrust::device_ptr<int> id_ptr(global_id);
    thrust::device_ptr<int> cell_id_ptr(cell_id);
    thrust::device_ptr<int> local_id_ptr(local_id);
    thrust::device_ptr<int> id_sorted_ptr(global_id_sortted);
    thrust::device_ptr<int> valid_ptr(d_valid);
    // 创建临时数组（分配长度为 N）
    thrust::device_vector<double3> new_pos(N);
    thrust::device_vector<double3> new_vel(N);
    thrust::device_vector<double>  new_mass(N);
    thrust::device_vector<int>     new_id(N);
    thrust::device_vector<int>     new_cell_id(N);
    thrust::device_vector<int>     new_local_id(N);
    thrust::device_vector<int>     new_sorted_id(N);

    // 执行 copy_if 到新数组
    auto new_end = thrust::copy_if(pos_ptr, pos_ptr + N, valid_ptr, new_pos.begin(), thrust::identity<int>());
    int N_new = new_end - new_pos.begin();

    thrust::copy_if(vel_ptr, vel_ptr + N, valid_ptr, new_vel.begin(), thrust::identity<int>());
    thrust::copy_if(mass_ptr, mass_ptr + N, valid_ptr, new_mass.begin(), thrust::identity<int>());
    thrust::copy_if(id_ptr, id_ptr + N, valid_ptr, new_id.begin(), thrust::identity<int>());
    thrust::copy_if(cell_id_ptr, cell_id_ptr + N, valid_ptr, new_cell_id.begin(), thrust::identity<int>());
    thrust::copy_if(local_id_ptr, local_id_ptr + N, valid_ptr, new_local_id.begin(), thrust::identity<int>());
    thrust::copy_if(id_sorted_ptr, id_sorted_ptr + N, valid_ptr, new_sorted_id.begin(), thrust::identity<int>());

    // 拷贝回原始数组
    thrust::copy(new_pos.begin(), new_pos.begin() + N_new, pos_ptr);
    thrust::copy(new_vel.begin(), new_vel.begin() + N_new, vel_ptr);
    thrust::copy(new_mass.begin(), new_mass.begin() + N_new, mass_ptr);
    thrust::copy(new_id.begin(), new_id.begin() + N_new, id_ptr);
    thrust::copy(new_cell_id.begin(), new_cell_id.begin() + N_new, cell_id_ptr);
    thrust::copy(new_local_id.begin(), new_local_id.begin() + N_new, local_id_ptr);
    thrust::copy(new_sorted_id.begin(), new_sorted_id.begin() + N_new, id_sorted_ptr);

    std::cout <<"run out : "<< N - N_new << " particles"<<std::endl;
    // 更新粒子数
    N = N_new;
}

void GPUParticles::Sort(const int* d_particleStartIndex)
{	
	int blockSize = 128;
	int numBlocks = (N + blockSize - 1) / blockSize;
	GPUParticleKernels::sortParticles<<<numBlocks, blockSize>>>(cell_id, local_id, global_id, global_id_sortted, d_particleStartIndex, N);
}

void GPUParticles::Injet()
{
    double JetLength = V_jet * tau;
    double JetVolume = JetLength * L2 * L3;
    size_t JetParticleNum = ((JetVolume * Rho / mass) / Fn);
    int required {N + JetParticleNum};
    if(required >= m_Capacity){
        int newCapacity {1.2 * required};
        ResizeStorage(newCapacity);
        std::cout <<"The capacity is not enough And has been changed to : "<<m_Capacity<<std::endl;
    }
    hipDeviceSynchronize();
    int blockSize = 128;
    int numBlocks = (JetParticleNum + blockSize - 1) / blockSize;
    // reset injected counter
    hipMemset(d_injectedCounter, 0, sizeof(int));
    GPUParticleKernels::InjectParticles<<<numBlocks, blockSize>>>(d_pos, d_vel,
                                                                    global_id, N, 
                                                                JetParticleNum, d_injectedCounter);
    N += JetParticleNum;

    std::cout << "Injected : " << JetParticleNum << " Particles"<<std::endl;
}

void GPUParticles::ResizeStorage(const int &newCapacity)
{
    if (newCapacity <= m_Capacity) return;  // 不需要扩容

    // 分配新内存
    double3* new_d_pos;
    double3* new_d_vel;
    double*  new_d_mass;
    int*     new_d_id;
    int*     new_d_cell_id;
    int*     new_d_local_id;
    int*     new_d_id_sorted;

    hipMalloc(&new_d_pos, sizeof(double3) * newCapacity);
    hipMalloc(&new_d_vel, sizeof(double3) * newCapacity);
    hipMalloc(&new_d_mass, sizeof(double) * newCapacity);
    hipMalloc(&new_d_id, sizeof(int) * newCapacity);
    hipMalloc(&new_d_cell_id, sizeof(int) * newCapacity);
    hipMalloc(&new_d_local_id, sizeof(int) * newCapacity);
    hipMalloc(&new_d_id_sorted, sizeof(int) * newCapacity);


    // 拷贝旧数据
    hipMemcpy(new_d_pos, d_pos, sizeof(double3) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_vel, d_vel, sizeof(double3) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_mass, d_mass, sizeof(double) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_id, global_id, sizeof(int) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_cell_id, cell_id, sizeof(int) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_local_id, local_id, sizeof(int) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_id_sorted, global_id_sortted, sizeof(int) * N, hipMemcpyDeviceToDevice);

    // 释放旧内存
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(d_mass);
    // hipFree(d_valid);
    hipFree(global_id);
    hipFree(cell_id);
    hipFree(local_id);
    hipFree(global_id_sortted);

    // 更新指针与容量
    d_pos = new_d_pos;
    d_vel = new_d_vel;
    d_mass = new_d_mass;
    global_id = new_d_id;
    cell_id = new_d_cell_id;
    local_id = new_d_local_id;
    global_id_sortted = new_d_id_sorted;
    m_Capacity = newCapacity;
}

__global__ void GPUParticleKernels::moveParticles(double3* pos,
                               double3* vel,
                               int N, double dt, const Boundary* d_boundaries,
                               const int* d_ifCut, const GPUSegment* d_Segments, const int* CellID, int* d_valid) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N && d_valid[i] != 0) 
        return;
    
    // 读取到register memory
    auto local_pos = pos[i];
    auto local_vel = vel[i];
    double d_Vstd {sqrt(2 * boltz * T / mass)};
    hiprandState localState;
    hiprand_init(clock64() + 1234 * threadIdx.x + blockIdx.x, 0, 0, &localState);

    local_pos.x += local_vel.x * dt;
    local_pos.y += local_vel.y * dt;
    // local_pos.z += local_vel.z * dt;

    /*========与流场内物体相碰撞=========*/
    int cellid {CellID[i]};
    bool ifcut {d_ifCut[cellid]};
    if(ifcut){
        auto segment {d_Segments[cellid]};
        if(segment.isHit(local_pos)){
            segment.Reflect(local_pos, local_vel, dt, localState);
        }
    }
    
    /*============边界条件=============*/
    if(local_pos.x < 0){
        // auto boundary = d_boundaries[0];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        // GPUBoundary::PeriodicBoundary::apply(local_pos, boundary.point, boundary.normal);
        

        // double x = hiprand_uniform(&localState) * (V_jet * tau);
        // double y = hiprand_uniform(&localState) * L2;
        // double z = hiprand_uniform(&localState) * L3;
        // auto velocity {GPURandomKernels::MaxwellDistribution(d_Vstd, localState)};

        // local_pos.x = x;
        // local_pos.y = y;
        // local_vel.z = z;
        // local_vel = velocity;
        // local_vel.x += V_jet;

        d_valid[i] = 0;
    }
    if(local_pos.x > L1){
        // auto boundary = d_boundaries[1];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        // GPUBoundary::PeriodicBoundary::apply(local_pos, boundary.point, boundary.normal);

        // double x = hiprand_uniform(&localState) * (V_jet * tau);
        // double y = hiprand_uniform(&localState) * L2;
        // double z = hiprand_uniform(&localState) * L3;
        // auto velocity {GPURandomKernels::MaxwellDistribution(d_Vstd, localState)};

        // local_pos.x = x;
        // local_pos.y = y;
        // local_vel.z = z;
        // local_vel = velocity;
        // local_vel.x += V_jet;
        d_valid[i] = 0;
    }
    if(local_pos.y < 0){
        // auto boundary = d_boundaries[2];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        // auto dt_ac = (local_pos.y) / local_vel.y;
        // auto rand1 {hiprand_uniform(&localState)};
        // auto a1 = sqrt(-log(rand1)) * d_Vstd;
        // auto rand2 {hiprand_uniform(&localState)};
        // auto a2 = 2 * M_PI * rand2;
        // local_vel.x = sin(a2) * a1;
        // local_vel.z = cos(a2) * a1;
        
        // double rand3 = hiprand_uniform(&localState);
        // local_vel.y = fabs(sqrt(-log(rand3)) * d_Vstd);
        // local_pos.y = fabs(local_vel.y * dt_ac);

        // double x = hiprand_uniform(&localState) * (V_jet * tau);
        // double y = hiprand_uniform(&localState) * L2;
        // double z = hiprand_uniform(&localState) * L3;
        // auto velocity {GPURandomKernels::MaxwellDistribution(d_Vstd, localState)};

        // local_pos.x = x;
        // local_pos.y = y;
        // local_vel = velocity;
        // local_vel.x += V_jet;

        d_valid[i] = 0;
    }
    if (local_pos.y > L2) {
        // auto boundary = d_boundaries[3];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        // double dt_ac = (local_pos.y - L2) / local_vel.y; // old vy > 0
        // // 采样一个新的速度（热壁反射 + 驱动盖速度）
        // double rand1 = hiprand_uniform(&localState);
        // double a1 = sqrt(-log(rand1)) * d_Vstd;  // 热壁麦克斯韦速度采样
        // double rand2 = hiprand_uniform(&localState);
        // double a2 = 2.0 * M_PI * rand2;
    
        // local_vel.x = a1 * cos(a2) + 300.0;  // 加上顶盖速度
        // local_vel.z = a1 * sin(a2);
    
        // double rand3 = hiprand_uniform(&localState);
        // local_vel.y = -sqrt(-log(rand3)) * d_Vstd;  // 反向的vy
        // local_pos.y = L2 - fabs(local_vel.y * dt_ac);

        // double x = hiprand_uniform(&localState) * (V_jet * tau);
        // double y = hiprand_uniform(&localState) * L2;
        // double z = hiprand_uniform(&localState) * L3;
        // auto velocity {GPURandomKernels::MaxwellDistribution(d_Vstd, localState)};

        // local_pos.x = x;
        // local_pos.y = y;
        // local_vel = velocity;
        // local_vel.x += V_jet;

        d_valid[i] = 0;
    }
    
    // if(local_pos.z < 0){
    //     local_pos.z = fmod(local_pos.z, L3) + L3;
    // }

    // if(local_pos.z > L3){
    //     local_pos.z = fmod(local_pos.z, L3);
    // }

    // 写回global memory
    pos[i] = local_pos;
    vel[i] = local_vel;
}

__global__ void GPUParticleKernels::sortParticles(const int *cell_id, const int *local_id, const int *global_id, int *global_id_sortted, const int *d_particleStartIndex, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) 
        return;
	
	int sorted_global_id = d_particleStartIndex[cell_id[i]] + local_id[i];
	global_id_sortted[sorted_global_id] = i;
}

__global__ void GPUParticleKernels::InjectParticles(
    double3* d_pos,
    double3* d_vel,
    int*     d_globalID,
    int N,                      // 已有的粒子数
    int maxInject,              // 注入的粒子数
    int* d_injectedCounter      // 原子变量，记录已注入粒子数（决定 global_id）
) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= maxInject) return;

    hiprandState localState;
    hiprand_init(clock64() + 1234 * threadIdx.x + blockIdx.x, 0, 0, &localState);

    double Vstd {sqrt(2 * boltz * T / mass)};
    // ----------------- 采样位置 -----------------
    double x = hiprand_uniform(&localState) * (V_jet * tau);
    double y = hiprand_uniform(&localState) * L2;
    double z = hiprand_uniform(&localState) * L3;

    // ----------------- 采样速度（Maxwell） -----------------
    // double a1 = sqrt(-log(hiprand_uniform(&localState))) * Vstd;
    // double a2 = 2.0 * M_PI * hiprand_uniform(&localState);
    // double vx = a1 * cos(a2);
    // double vy = a1 * sin(a2);
    // double vz = sqrt(-log(hiprand_uniform(&localState))) * Vstd;
    // vx += V_jet;  // 偏移速度分量（喷射方向）

    auto velocity {GPURandomKernels::MaxwellDistribution(Vstd, localState)};
    velocity.x += V_jet;
    // velocity.y += 0.5 * V_jet;

    // ----------------- 原子分配 global_id -----------------
    int index = atomicAdd(d_injectedCounter, 1);  // 分配当前粒子在数组中的 index（全局 ID）

    if (index >= maxInject) return;  // 避免越界写入
    index += N;
    // ----------------- 写入粒子信息 -----------------
    d_pos[index] = make_double3(x, y, z);
    // d_vel[index] = make_double3(vx, vy, vz);
    d_vel[index] = velocity;
    d_globalID[index] = index;
}


__device__ double3 GPURandomKernels::MaxwellDistribution(const double& Vstd, hiprandState& localState)
{
    double rd1 = hiprand_uniform(&localState);
    double rd2 = hiprand_uniform(&localState);
    double u = sqrt(-log(rd1)) * sin(2.0 * M_PI * rd2) * Vstd;

    rd1 = hiprand_uniform(&localState);
    rd2 = hiprand_uniform(&localState);
    double v = sqrt(-log(rd1)) * sin(2.0 * M_PI * rd2) * Vstd;

    rd1 = hiprand_uniform(&localState);
    rd2 = hiprand_uniform(&localState);
    double w = sqrt(-log(rd1)) * sin(2.0 * M_PI * rd2) * Vstd;

    return make_double3(u, v, w);
}