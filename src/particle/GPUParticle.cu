#include "hip/hip_runtime.h"
#include "Particle.cuh"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>


GPUParticles::GPUParticles(const int &particleNum) : N(particleNum)
{   
    size_t sizedoubles {N * sizeof(double)};   
    size_t sizedouble3s {N * sizeof(double3)};
    size_t sizeints {N * sizeof(int)};
    hipMalloc((void**)&d_mass, sizedoubles);
    hipMalloc((void**)&d_pos, sizedouble3s);
    hipMalloc((void**)&d_vel, sizedouble3s);
    hipMalloc((void**)&global_id, sizeints);
	hipMalloc((void**)&global_id_sortted, sizeints);
    hipMalloc((void**)&cell_id, sizeints);
    hipMalloc((void**)&local_id, sizeints);
    hipMalloc((void**)&d_injectedCounter, sizeof(int));

}

GPUParticles::~GPUParticles()
{
    hipFree(d_mass);
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(global_id);
	hipFree(global_id_sortted);
    hipFree(local_id);
    hipFree(cell_id);
    hipFree(d_injectedCounter);
}

void GPUParticles::UploadFromHost(const double* h_mass,
    const double3* h_pos,
    const double3* h_vel, 
    const int* h_global_id, const int* h_local_id, const int* h_cell_id)
{
    size_t sizedoubles {N * sizeof(double)};   
    size_t sizedouble3s {N * sizeof(double3)};
    size_t sizeints {N * sizeof(int)};
    hipMemcpy(d_mass, h_mass, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_pos, h_pos, sizedouble3s, hipMemcpyHostToDevice);
    hipMemcpy(d_vel, h_vel, sizedouble3s, hipMemcpyHostToDevice);

    hipMemcpy(global_id, h_global_id, sizeints, hipMemcpyHostToDevice);
    hipMemcpy(local_id, h_local_id, sizeints, hipMemcpyHostToDevice);
    hipMemcpy(cell_id, h_cell_id, sizeints, hipMemcpyHostToDevice);
}

void GPUParticles::Move(const double &dt, const double &blockSize, const Boundary* d_boundaries,
    const int* d_ifCut, const GPUSegment* d_Segments)
{
    int numBlocks = (N + blockSize - 1) / blockSize;
    GPUParticleKernels::moveParticles<<<numBlocks, blockSize>>>(d_pos,
                                          d_vel,
                                          N, dt, d_boundaries,
                                          d_ifCut, d_Segments, cell_id);
    hipDeviceSynchronize();
}

void GPUParticles::Sort(const int* d_particleStartIndex)
{	
	int blockSize = 128;
	int numBlocks = (N + blockSize - 1) / blockSize;
	GPUParticleKernels::sortParticles<<<numBlocks, blockSize>>>(cell_id, local_id, global_id, global_id_sortted, d_particleStartIndex, N);
}

void GPUParticles::Injet()
{
    double JetLength = V_jet * tau;
    double JetVolume = JetLength * L2 * L3;
    size_t JetParticleNum = ((JetVolume * Rho / mass) / Fn);
    int required {N + JetParticleNum};
    if(required >= m_Capacity){
        int newCapacity {1.2 * required};
        ResizeStorage(newCapacity);
        std::cout <<"The capacity is not enough And has been changed to : "<<m_Capacity<<std::endl;
    }
    hipDeviceSynchronize();
    int blockSize = 128;
    int numBlocks = (JetParticleNum + blockSize - 1) / blockSize;
    // reset injected counter
    hipMemset(d_injectedCounter, 0, sizeof(int));
    GPUParticleKernels::InjectParticles<<<numBlocks, blockSize>>>(d_pos, d_vel,
                                                                    global_id, N, 
                                                                JetParticleNum, d_injectedCounter);
    N += JetParticleNum;

    std::cout << "Injected : " << JetParticleNum << " Particles"<<std::endl;
}

void GPUParticles::ResizeStorage(const int &newCapacity)
{
    if (newCapacity <= m_Capacity) return;  // 不需要扩容

    // 分配新内存
    double3* new_d_pos;
    double3* new_d_vel;
    double*  new_d_mass;
    int*     new_d_id;
    int*     new_d_cell_id;
    int*     new_d_local_id;
    int*     new_d_id_sorted;

    hipMalloc(&new_d_pos, sizeof(double3) * newCapacity);
    hipMalloc(&new_d_vel, sizeof(double3) * newCapacity);
    hipMalloc(&new_d_mass, sizeof(double) * newCapacity);
    hipMalloc(&new_d_id, sizeof(int) * newCapacity);
    hipMalloc(&new_d_cell_id, sizeof(int) * newCapacity);
    hipMalloc(&new_d_local_id, sizeof(int) * newCapacity);
    hipMalloc(&new_d_id_sorted, sizeof(int) * newCapacity);

    // 拷贝旧数据
    hipMemcpy(new_d_pos, d_pos, sizeof(double3) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_vel, d_vel, sizeof(double3) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_mass, d_mass, sizeof(double) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_id, global_id, sizeof(int) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_cell_id, cell_id, sizeof(int) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_local_id, local_id, sizeof(int) * N, hipMemcpyDeviceToDevice);
    hipMemcpy(new_d_id_sorted, global_id_sortted, sizeof(int) * N, hipMemcpyDeviceToDevice);

    // 释放旧内存
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(d_mass);
    hipFree(global_id);
    hipFree(cell_id);
    hipFree(local_id);
    hipFree(global_id_sortted);

    // 更新指针与容量
    d_pos = new_d_pos;
    d_vel = new_d_vel;
    d_mass = new_d_mass;
    global_id = new_d_id;
    cell_id = new_d_cell_id;
    local_id = new_d_local_id;
    global_id_sortted = new_d_id_sorted;
    m_Capacity = newCapacity;
}

__global__ void GPUParticleKernels::moveParticles(double3* pos,
                               double3* vel,
                               int N, double dt, const Boundary* d_boundaries,
                               const int* d_ifCut, const GPUSegment* d_Segments, const int* CellID) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) 
        return;
    
    // 读取到register memory
    auto local_pos = pos[i];
    auto local_vel = vel[i];
    double d_Vstd {sqrt(2 * boltz * T / mass)};
    hiprandState localState;
    hiprand_init(clock64() + 1234 * threadIdx.x + blockIdx.x, 0, 0, &localState);

    local_pos.x += local_vel.x * dt;
    local_pos.y += local_vel.y * dt;
    // local_pos.z += local_vel.z * dt;

    /*========与流场内物体相碰撞=========*/
    int cellid {CellID[i]};
    bool ifcut {d_ifCut[cellid]};
    if(ifcut){
        auto segment {d_Segments[cellid]};
        if(segment.isHit(local_pos)){
            segment.Reflect(local_pos, local_vel, dt);
        }
    }
    
    /*============边界条件=============*/
    if(local_pos.x < 0){
        auto boundary = d_boundaries[0];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        // GPUBoundary::PeriodicBoundary::apply(local_pos, boundary.point, boundary.normal);
    }
    if(local_pos.x > L1){
        auto boundary = d_boundaries[1];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        // GPUBoundary::PeriodicBoundary::apply(local_pos, boundary.point, boundary.normal);
    }
    if(local_pos.y < 0){
        // auto boundary = d_boundaries[2];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        auto dt_ac = (local_pos.y) / local_vel.y;
        auto rand1 {hiprand_uniform(&localState)};
        auto a1 = sqrt(-log(rand1)) * d_Vstd;
        auto rand2 {hiprand_uniform(&localState)};
        auto a2 = 2 * M_PI * rand2;
        local_vel.x = sin(a2) * a1;
        local_vel.z = cos(a2) * a1;
        
        double rand3 = hiprand_uniform(&localState);
        local_vel.y = fabs(sqrt(-log(rand3)) * d_Vstd);
        local_pos.y = fabs(local_vel.y * dt_ac);
    }
    if (local_pos.y > L2) {
        // auto boundary = d_boundaries[3];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        double dt_ac = (local_pos.y - L2) / local_vel.y; // old vy > 0
    
        // 采样一个新的速度（热壁反射 + 驱动盖速度）
        double rand1 = hiprand_uniform(&localState);
        double a1 = sqrt(-log(rand1)) * d_Vstd;  // 热壁麦克斯韦速度采样
        double rand2 = hiprand_uniform(&localState);
        double a2 = 2.0 * M_PI * rand2;
    
        local_vel.x = a1 * cos(a2) + 000.0;  // 加上顶盖速度
        local_vel.z = a1 * sin(a2);
    
        double rand3 = hiprand_uniform(&localState);
        local_vel.y = -sqrt(-log(rand3)) * d_Vstd;  // 反向的vy
        local_pos.y = L2 - fabs(local_vel.y * dt_ac);
    }
    
    // if(local_pos.z < 0){
    //     local_pos.z = fmod(local_pos.z, L3) + L3;
    // }

    // if(local_pos.z > L3){
    //     local_pos.z = fmod(local_pos.z, L3);
    // }

    // 写回global memory
    pos[i] = local_pos;
    vel[i] = local_vel;
}

__global__ void GPUParticleKernels::sortParticles(const int *cell_id, const int *local_id, const int *global_id, int *global_id_sortted, const int *d_particleStartIndex, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) 
        return;
	
	int sorted_global_id = d_particleStartIndex[cell_id[i]] + local_id[i];
	global_id_sortted[sorted_global_id] = global_id[i];
}

__global__ void GPUParticleKernels::InjectParticles(
    double3* d_pos,
    double3* d_vel,
    int*     d_globalID,
    int N,                      // 已有的粒子数
    int maxInject,              // 注入的粒子数
    int* d_injectedCounter      // 原子变量，记录已注入粒子数（决定 global_id）
) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= maxInject) return;

    hiprandState localState;
    hiprand_init(clock64() + 1234 * threadIdx.x + blockIdx.x, 0, 0, &localState);

    double Vstd {sqrt(2 * boltz * T / mass)};
    // ----------------- 采样位置 -----------------
    double x = hiprand_uniform(&localState) * (V_jet * tau);
    double y = hiprand_uniform(&localState) * L2;
    double z = hiprand_uniform(&localState) * L3;

    // ----------------- 采样速度（Maxwell） -----------------
    double a1 = sqrt(-log(hiprand_uniform(&localState))) * Vstd;
    double a2 = 2.0 * M_PI * hiprand_uniform(&localState);
    double vx = a1 * cos(a2);
    double vy = a1 * sin(a2);
    double vz = sqrt(-log(hiprand_uniform(&localState))) * Vstd;

    vx += V_jet;  // 偏移速度分量（喷射方向）

    // ----------------- 原子分配 global_id -----------------
    int index = atomicAdd(d_injectedCounter, 1);  // 分配当前粒子在数组中的 index（全局 ID）

    if (index >= maxInject) return;  // 避免越界写入
    index += N;
    // ----------------- 写入粒子信息 -----------------
    d_pos[index] = make_double3(x, y, z);
    d_vel[index] = make_double3(vx, vy, vz);
    d_globalID[index] = index;
}
