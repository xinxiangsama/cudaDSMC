#include "hip/hip_runtime.h"
#include "Particle.cuh"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <hiprand/hiprand_kernel.h>



extern __constant__ double d_Unidx;
extern __constant__ double d_Unidy;
extern __constant__ double d_Unidz;
extern __constant__ double d_Vstd;

GPUParticles::GPUParticles(const int &particleNum) : N(particleNum)
{   
    size_t sizedoubles {N * sizeof(double)};   
    size_t sizeints {N * sizeof(int)};
    hipMalloc((void**)&d_mass, sizedoubles);
    hipMalloc((void**)&d_pos_x, sizedoubles);
    hipMalloc((void**)&d_pos_y, sizedoubles);
    hipMalloc((void**)&d_pos_z, sizedoubles);
    hipMalloc((void**)&d_vel_x, sizedoubles);
    hipMalloc((void**)&d_vel_y, sizedoubles);
    hipMalloc((void**)&d_vel_z, sizedoubles);  
    hipMalloc((void**)&global_id, sizeints);
	hipMalloc((void**)&global_id_sortted, sizeints);
    hipMalloc((void**)&cell_id, sizeints);
    hipMalloc((void**)&local_id, sizeints);

}

GPUParticles::~GPUParticles()
{
    hipFree(d_mass);
    hipFree(d_pos_x);
    hipFree(d_pos_y);
    hipFree(d_pos_z);
    hipFree(d_vel_x);
    hipFree(d_vel_y);
    hipFree(d_vel_z);
    hipFree(global_id);
	hipFree(global_id_sortted);
    hipFree(local_id);
    hipFree(cell_id);
}

void GPUParticles::UploadFromHost(const double* h_mass,
    const double* h_pos_x, const double* h_pos_y, const double* h_pos_z,
    const double* h_vel_x, const double* h_vel_y, const double* h_vel_z, const int* h_global_id, const int* h_local_id, const int* h_cell_id)
{
    size_t sizedoubles {N * sizeof(double)};
    size_t sizeints {N * sizeof(int)};
    hipMemcpy(d_mass, h_mass, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_pos_x, h_pos_x, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_pos_y, h_pos_y, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_pos_z, h_pos_z, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_vel_x, h_vel_x, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_vel_y, h_vel_y, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_vel_z, h_vel_z, sizedoubles, hipMemcpyHostToDevice);

    hipMemcpy(global_id, h_global_id, sizeints, hipMemcpyHostToDevice);
    hipMemcpy(local_id, h_local_id, sizeints, hipMemcpyHostToDevice);
    hipMemcpy(cell_id, h_cell_id, sizeints, hipMemcpyHostToDevice);
}

void GPUParticles::Move(const double &dt, const double &blockSize, const Boundary* d_boundaries)
{
    int numBlocks = (N + blockSize - 1) / blockSize;
    GPUParticleKernels::moveParticles<<<numBlocks, blockSize>>>(d_pos_x, d_pos_y, d_pos_z,
                                          d_vel_x, d_vel_y, d_vel_z,
                                          N, dt, d_boundaries);
    hipDeviceSynchronize();
}

void GPUParticles::Sort(const int* d_particleStartIndex)
{	
	int blockSize = 128;
	int numBlocks = (N + blockSize - 1) / blockSize;
	GPUParticleKernels::sortParticles<<<numBlocks, blockSize>>>(cell_id, local_id, global_id, global_id_sortted, d_particleStartIndex, N);
}

__global__ void GPUParticleKernels::moveParticles(double* pos_x, double* pos_y, double* pos_z,
                               double* vel_x, double* vel_y, double* vel_z,
                               int N, double dt, const Boundary* d_boundaries) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) 
        return;

    pos_x[i] += vel_x[i] * dt;
    pos_y[i] += vel_y[i] * dt;  

    /*============边界条件=============*/
    /*x方向*/
    if(pos_x[i] < 0){
		// pos_x[i] = fmod(pos_x[i], L1) + L1;
        auto pos = make_double3(pos_x[i], pos_y[i], pos_z[i]);
        // auto point = make_double3(0, 0.5 * L2, 0.5 * L3);
        // GPUBoundary::PeriodicBoundary::apply(pos, point, make_double3(1, 0, 0));
        GPUBoundary::PeriodicBoundary::apply(pos, d_boundaries[0].point, d_boundaries[0].normal);
        pos_x[i] = pos.x;
	}

	if(pos_x[i] > L1){
		// pos_x[i] = fmod(pos_x[i], L1);
        auto pos = make_double3(pos_x[i], pos_y[i], pos_z[i]);
        // auto point = make_double3(L1, 0.5 * L2, 0.5 * L3);
        // GPUBoundary::PeriodicBoundary::apply(pos, point, make_double3(-1, 0, 0));
        GPUBoundary::PeriodicBoundary::apply(pos, d_boundaries[1].point, d_boundaries[1].normal);
        pos_x[i] = pos.x;
	}
	/*y方向*/
	if (pos_y[i] < 0) {
		// pos_y[i] = fmod(pos_y[i], L2) + L2;
        auto pos = make_double3(pos_x[i], pos_y[i], pos_z[i]);
        auto vel = make_double3(vel_x[i], vel_y[i], vel_z[i]);
        auto point = make_double3(0.5 * L1, 0.0, 0.5 * L3);
        GPUBoundary::WallBoundary::apply(pos, vel, point, make_double3(0, 1, 0));
        pos_y[i] = pos.y;
        vel_y[i] = vel.y;
	}

	if (pos_y[i] > L2) {
		// pos_y[i] = fmod(pos_y[i], L2);
        auto pos = make_double3(pos_x[i], pos_y[i], pos_z[i]);
        auto vel = make_double3(vel_x[i], vel_y[i], vel_z[i]);
        auto point = make_double3(0.5 * L1, L2, 0.5 * L3);
        GPUBoundary::WallBoundary::apply(pos, vel, point, make_double3(0, -1, 0));
        pos_y[i] = pos.y;
        vel_y[i] = vel.y;
        vel_x[i] = 300;
	}

    // pos_z[i] += vel_z[i] * dt;                       
    // // 检查所有边界
    // for (int b = 0; b < 6; ++b) {
    //     const Boundary& boundary = d_boundaries[b];
    //     if (GPUBoundary::isHit(pos, boundary.point, boundary.normal)) {
    //         // if (boundary.type == GPUBoundary::BoundaryType::PERIODIC)
    //             // GPUBoundary::PeriodicBoundary::apply(pos, boundary.point, boundary.normal);
    //     }
    // }

}

__global__ void GPUParticleKernels::sortParticles(const int *cell_id, const int *local_id, const int *global_id, int *global_id_sortted, const int *d_particleStartIndex, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) 
        return;
	
	int sorted_global_id = d_particleStartIndex[cell_id[i]] + local_id[i];
	global_id_sortted[sorted_global_id] = global_id[i];
}
