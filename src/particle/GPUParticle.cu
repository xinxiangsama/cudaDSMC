#include "hip/hip_runtime.h"
#include "Particle.cuh"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>


GPUParticles::GPUParticles(const int &particleNum) : N(particleNum)
{   
    size_t sizedoubles {N * sizeof(double)};   
    size_t sizedouble3s {N * sizeof(double3)};
    size_t sizeints {N * sizeof(int)};
    hipMalloc((void**)&d_mass, sizedoubles);
    hipMalloc((void**)&d_pos, sizedouble3s);
    hipMalloc((void**)&d_vel, sizedouble3s);
    hipMalloc((void**)&global_id, sizeints);
	hipMalloc((void**)&global_id_sortted, sizeints);
    hipMalloc((void**)&cell_id, sizeints);
    hipMalloc((void**)&local_id, sizeints);

}

GPUParticles::~GPUParticles()
{
    hipFree(d_mass);
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(global_id);
	hipFree(global_id_sortted);
    hipFree(local_id);
    hipFree(cell_id);
}

void GPUParticles::UploadFromHost(const double* h_mass,
    const double3* h_pos,
    const double3* h_vel, 
    const int* h_global_id, const int* h_local_id, const int* h_cell_id)
{
    size_t sizedoubles {N * sizeof(double)};   
    size_t sizedouble3s {N * sizeof(double3)};
    size_t sizeints {N * sizeof(int)};
    hipMemcpy(d_mass, h_mass, sizedoubles, hipMemcpyHostToDevice);
    hipMemcpy(d_pos, h_pos, sizedouble3s, hipMemcpyHostToDevice);
    hipMemcpy(d_vel, h_vel, sizedouble3s, hipMemcpyHostToDevice);

    hipMemcpy(global_id, h_global_id, sizeints, hipMemcpyHostToDevice);
    hipMemcpy(local_id, h_local_id, sizeints, hipMemcpyHostToDevice);
    hipMemcpy(cell_id, h_cell_id, sizeints, hipMemcpyHostToDevice);
}

void GPUParticles::Move(const double &dt, const double &blockSize, const Boundary* d_boundaries,
    const int* d_ifCut, const GPUSegment* d_Segments)
{
    int numBlocks = (N + blockSize - 1) / blockSize;
    GPUParticleKernels::moveParticles<<<numBlocks, blockSize>>>(d_pos,
                                          d_vel,
                                          N, dt, d_boundaries,
                                          d_ifCut, d_Segments, cell_id);
    hipDeviceSynchronize();
}

void GPUParticles::Sort(const int* d_particleStartIndex)
{	
	int blockSize = 128;
	int numBlocks = (N + blockSize - 1) / blockSize;
	GPUParticleKernels::sortParticles<<<numBlocks, blockSize>>>(cell_id, local_id, global_id, global_id_sortted, d_particleStartIndex, N);
}

__global__ void GPUParticleKernels::moveParticles(double3* pos,
                               double3* vel,
                               int N, double dt, const Boundary* d_boundaries,
                               const int* d_ifCut, const GPUSegment* d_Segments, const int* CellID) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) 
        return;
    
    // 读取到register memory
    auto local_pos = pos[i];
    auto local_vel = vel[i];
    double d_Vstd {sqrt(2 * boltz * T / mass)};
    hiprandState localState;
    hiprand_init(clock64() + 1234 * threadIdx.x + blockIdx.x, 0, 0, &localState);

    local_pos.x += local_vel.x * dt;
    local_pos.y += local_vel.y * dt;
    // local_pos.z += local_vel.z * dt;

    /*========与流场内物体相碰撞=========*/
    int cellid {CellID[i]};
    bool ifcut {d_ifCut[cellid]};
    if(ifcut){
        auto segment {d_Segments[cellid]};
        if(segment.isHit(local_pos)){
            segment.Reflect(local_pos, local_vel, dt);
        }
    }
    
    /*============边界条件=============*/
    if(local_pos.x < 0){
        auto boundary = d_boundaries[0];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        GPUBoundary::PeriodicBoundary::apply(local_pos, boundary.point, boundary.normal);
    }
    if(local_pos.x > L1){
        auto boundary = d_boundaries[1];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        GPUBoundary::PeriodicBoundary::apply(local_pos, boundary.point, boundary.normal);
    }
    if(local_pos.y < 0){
        // auto boundary = d_boundaries[2];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        auto dt_ac = (local_pos.y) / local_vel.y;
        auto rand1 {hiprand_uniform(&localState)};
        auto a1 = sqrt(-log(rand1)) * d_Vstd;
        auto rand2 {hiprand_uniform(&localState)};
        auto a2 = 2 * M_PI * rand2;
        local_vel.x = sin(a2) * a1;
        local_vel.z = cos(a2) * a1;
        
        double rand3 = hiprand_uniform(&localState);
        local_vel.y = fabs(sqrt(-log(rand3)) * d_Vstd);
        local_pos.y = fabs(local_vel.y * dt_ac);
    }
    if (local_pos.y > L2) {
        // auto boundary = d_boundaries[3];
        // GPUBoundary::WallBoundary::apply(local_pos, local_vel, boundary.point, boundary.normal);
        double dt_ac = (local_pos.y - L2) / local_vel.y; // old vy > 0
    
        // 采样一个新的速度（热壁反射 + 驱动盖速度）
        double rand1 = hiprand_uniform(&localState);
        double a1 = sqrt(-log(rand1)) * d_Vstd;  // 热壁麦克斯韦速度采样
        double rand2 = hiprand_uniform(&localState);
        double a2 = 2.0 * M_PI * rand2;
    
        local_vel.x = a1 * cos(a2) + 000.0;  // 加上顶盖速度
        local_vel.z = a1 * sin(a2);
    
        double rand3 = hiprand_uniform(&localState);
        local_vel.y = -sqrt(-log(rand3)) * d_Vstd;  // 反向的vy
        local_pos.y = L2 - fabs(local_vel.y * dt_ac);
    }
    
    // if(local_pos.z < 0){
    //     local_pos.z = fmod(local_pos.z, L3) + L3;
    // }

    // if(local_pos.z > L3){
    //     local_pos.z = fmod(local_pos.z, L3);
    // }

    // 写回global memory
    pos[i] = local_pos;
    vel[i] = local_vel;
}

__global__ void GPUParticleKernels::sortParticles(const int *cell_id, const int *local_id, const int *global_id, int *global_id_sortted, const int *d_particleStartIndex, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) 
        return;
	
	int sorted_global_id = d_particleStartIndex[cell_id[i]] + local_id[i];
	global_id_sortted[sorted_global_id] = global_id[i];
}
